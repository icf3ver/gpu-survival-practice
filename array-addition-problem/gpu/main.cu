
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int* a, int* b, int* c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
    return;
}

int main() {
	int a[] = {1,2,3};
	int b[] = {4,5,6};
	int c[sizeof(a) / sizeof(int)] = {0};
	// Creating pointers into GPU
	int* cudaA = 0;
	int* cudaB = 0;
	int* cudaC = 0;
	// Allocate memory in the GPU
	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));
	// Copy the vectors into the gpu
	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
	// Launch the kernel with one block and a number of threads equal to the size of the vectors
	vectorAdd <<<1, sizeof(a) / sizeof(a[0])>>> (cudaA, cudaB, cudaC);
	// Copy the result vector back to the host
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
	// Print the result
	for (int i = 0; i < sizeof(c) / sizeof(int); i++) {
		printf("c[%d] = %d\n", i, c[i]);
	}

	return 0;
}
